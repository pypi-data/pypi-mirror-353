#include "hip/hip_runtime.h"
#include <cmath>
#include <complex>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "gpu_wrapping.h"
// #include <sys/time.h>

#include <limits.h>

#include "stat_ops.h"
#include "stat_ops_device_functions.h"
#include "update_ops_cuda.h"
#include "util.cuh"
#include "util_func.h"
#include "util_type.h"

__constant__ GTYPE matrix_const_gpu[4];
__constant__ unsigned int num_pauli_op_gpu[4];
__constant__ ITYPE bit_mask_gpu[4];
__constant__ GTYPE PHASE_90ROT_gpu[4];

__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

inline __device__ double __shfl_down_double(
#ifdef __HIP_PLATFORM_AMD__
    double var, unsigned int srcLane, int width = 64) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width);
    a.y = __shfl_down(a.y, srcLane, width);
#else
    double var, unsigned int srcLane, int width = 32) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down_sync(0xffffffff, a.x, srcLane, width);
    a.y = __shfl_down_sync(0xffffffff, a.y, srcLane, width);
#endif
    return *reinterpret_cast<double*>(&a);
}

inline __device__ double __shfl_xor_double(
#ifdef __HIP_PLATFORM_AMD__
    double var, unsigned int srcLane, int width = 64) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_xor(a.x, srcLane, width);
    a.y = __shfl_xor(a.y, srcLane, width);
#else
    double var, unsigned int srcLane, int width = 32) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_xor_sync(0xffffffff, a.x, srcLane, width);
    a.y = __shfl_xor_sync(0xffffffff, a.y, srcLane, width);
#endif
    return *reinterpret_cast<double*>(&a);
}

inline __device__ double warpReduceSum_double(double val) {
#pragma unroll
    for (int offset = (warpSize >> 1); offset > 0; offset >>= 1)
#ifdef __HIP_PLATFORM_AMD__
        val += __shfl_down(val, offset);
#else
        val += __shfl_down_sync(0xffffffff, val, offset);
#endif
    return val;
}

inline __device__ double warpAllReduceSum_double(double val) {
#ifdef __HIP_PLATFORM_AMD__
    val += __shfl_xor(val, 16);
    val += __shfl_xor(val, 8);
    val += __shfl_xor(val, 4);
    val += __shfl_xor(val, 2);
    val += __shfl_xor(val, 1);
#else
    val += __shfl_xor_sync(0xffffffff, val, 16);
    val += __shfl_xor_sync(0xffffffff, val, 8);
    val += __shfl_xor_sync(0xffffffff, val, 4);
    val += __shfl_xor_sync(0xffffffff, val, 2);
    val += __shfl_xor_sync(0xffffffff, val, 1);
#endif
    return val;
}

__global__ void state_norm_squared_gpu(double* ret, GTYPE* state, ITYPE dim) {
    double sum = double(0.0);
    GTYPE tmp;
    ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (ITYPE i = idx; i < dim; i += blockDim.x * gridDim.x) {
        tmp = state[i];
        sum += tmp.x * tmp.x + tmp.y * tmp.y;
    }
    sum = warpReduceSum_double(sum);

    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(ret, sum);
    }
}

__host__ double state_norm_squared_cublas_host(void* state, ITYPE dim) {
    gpublasStatus_t status;
    gpublasHandle_t handle;
    double norm;
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

    /* Initialize CUBLAS */
    status = gpublasCreate(&handle);
    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    status =
        gpublasDznrm2(handle, dim, (gpublasDoubleComplex*)state_gpu, 1, &norm);
    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! hipblasDznrm2 execution error.\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = gpublasDestroy(handle);

    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error\n");
        return EXIT_FAILURE;
    }

    state = reinterpret_cast<void*>(state_gpu);
    return norm;
}

__host__ double state_norm_squared_host(
    void* state, ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    gpuError_t gpuStatus;
    double norm = 0.0;
    double* norm_gpu;
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

    checkGpuErrors(
        gpuMalloc((void**)&norm_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(norm_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);
    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(state_norm_squared_gpu);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;

    state_norm_squared_gpu<<<grid, block, 0, *gpu_stream>>>(
        norm_gpu, state_gpu, dim);

    // Check for any errors launching the kernel
    gpuStatus = gpuGetLastError();

    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&norm, norm_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);

    checkGpuErrors(gpuFree(norm_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return norm;
}

__global__ void measurement_distribution_entropy_gpu(
    double* ret, const GTYPE* state, ITYPE dim) {
    double sum = 0;
    const double eps = 1e-15;

    double prob;
    for (ITYPE index = blockIdx.x * blockDim.x + threadIdx.x; index < dim;
         index += blockDim.x * gridDim.x) {
        prob = gpuCabs(state[index]);
        prob = prob * prob;
        if (prob > eps) {
            sum += -1.0 * prob * log(prob);
        }
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

__host__ double measurement_distribution_entropy_host(
    void* state, ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    gpuError_t gpuStatus;
    double ent;
    double* ent_gpu;
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

    checkGpuErrors(
        gpuMalloc((void**)&ent_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ent_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);
    unsigned int max_block_size = get_block_size_to_maximize_occupancy(
        measurement_distribution_entropy_gpu);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;

    measurement_distribution_entropy_gpu<<<grid, block, 0, *gpu_stream>>>(
        ent_gpu, state_gpu, dim);

    // Check for any errors launching the kernel
    gpuStatus = gpuGetLastError();

    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&ent, ent_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);

    checkGpuErrors(gpuFree(ent_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);

    return ent;
}

__global__ void state_add_gpu(
    const GTYPE* state_added, GTYPE* state, ITYPE dim) {
    ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;

    // loop varaibles
    const ITYPE loop_dim = dim;
    if (state_index < loop_dim) {
        state[state_index] =
            gpuCadd(state[state_index], state_added[state_index]);
    }
}

__host__ void state_add_host(void* state_added, void* state, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    GTYPE* state_added_gpu = reinterpret_cast<GTYPE*>(state_added);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);

    ITYPE loop_dim = dim;

    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(state_add_gpu);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;

    state_add_gpu<<<grid, block, 0, *gpu_stream>>>(
        state_added_gpu, state_gpu, dim);

    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    state_added = reinterpret_cast<void*>(state_added_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
}

__global__ void state_multiply_gpu(const GTYPE coef, GTYPE* state, ITYPE dim) {
    ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;

    const ITYPE loop_dim = dim;
    if (state_index < loop_dim) {
        state[state_index] = gpuCmul(state[state_index], coef);
    }
}

__host__ void state_multiply_host(CPPCTYPE coef, void* state, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    ITYPE loop_dim = dim;

    GTYPE coef_gpu = make_gpuDoubleComplex(coef.real(), coef.imag());
    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(state_multiply_gpu);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;

    state_multiply_gpu<<<grid, block, 0, *gpu_stream>>>(
        coef_gpu, state_gpu, dim);

    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
}

__global__ void inner_product_gpu(
    GTYPE* ret, const GTYPE* psi, const GTYPE* phi, ITYPE dim) {
    GTYPE sum = make_gpuDoubleComplex(0.0, 0.0);
    for (ITYPE i = blockIdx.x * blockDim.x + threadIdx.x; i < dim;
         i += blockDim.x * gridDim.x) {
        sum = gpuCadd(sum, gpuCmul(gpuConj(psi[i]), phi[i]));
    }
    sum.x = warpReduceSum_double(sum.x);
    sum.y = warpReduceSum_double(sum.y);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0].x), sum.x);
        atomicAdd_double(&(ret[0].y), sum.y);
    }
}

/*
__host__ CPPCTYPE inner_product_cublas_host(const void *bra_state, const void
*ket_state, ITYPE dim) { const GTYPE* bra_state_gpu = reinterpret_cast<const
GTYPE*>(bra_state); const GTYPE* ket_state_gpu = reinterpret_cast<const
GTYPE*>(ket_state); hipblasStatus_t status; hipblasHandle_t handle; GTYPE ret_g;
    CPPCTYPE ret;

    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

        status = hipblasZdotc(handle, dim, bra_state_gpu, 1, ket_state_gpu, 1,
&ret_g); if (status != HIPBLAS_STATUS_SUCCESS) { fprintf(stderr, "!!!!
cublasZDotc execution error.\n"); return EXIT_FAILURE;
    }

    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error\n");
        return EXIT_FAILURE;
    }

        bra_state = reinterpret_cast<const void*>(bra_state_gpu);
        ket_state = reinterpret_cast<const void*>(ket_state_gpu);
    ret = CPPCTYPE(hipCreal(ret_g), hipCimag(ret_g));
        return ret;
}
*/

__host__ CPPCTYPE inner_product_cublas_host(const void* bra_state,
    const void* ket_state, ITYPE dim, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    const GTYPE* bra_state_gpu = reinterpret_cast<const GTYPE*>(bra_state);
    const GTYPE* ket_state_gpu = reinterpret_cast<const GTYPE*>(ket_state);
    gpublasStatus_t status;
    gpublasHandle_t handle;
    GTYPE ret_g;
    CPPCTYPE ret;

    /* Initialize CUBLAS */
    status = gpublasCreate(&handle);
    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    status = gpublasSetStream(handle, *gpu_stream);
    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! set cublas to cuda stream error\n");
        return EXIT_FAILURE;
    }

    status = gpublasZdotc(handle, dim, (gpublasDoubleComplex*)bra_state_gpu, 1,
        (gpublasDoubleComplex*)ket_state_gpu, 1, (gpublasDoubleComplex*)&ret_g);
    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! cublasZDotc execution error.\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = gpublasDestroy(handle);

    if (status != GPUBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error\n");
        return EXIT_FAILURE;
    }

    bra_state = reinterpret_cast<const void*>(bra_state_gpu);
    ket_state = reinterpret_cast<const void*>(ket_state_gpu);
    ret = CPPCTYPE(gpuCreal(ret_g), gpuCimag(ret_g));
    return ret;
}

__host__ CPPCTYPE inner_product_original_host(const void* bra_state,
    const void* ket_state, ITYPE dim, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    const GTYPE* bra_state_gpu = reinterpret_cast<const GTYPE*>(bra_state);
    const GTYPE* ket_state_gpu = reinterpret_cast<const GTYPE*>(ket_state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    gpuError_t gpuStatus;
    CPPCTYPE ret = CPPCTYPE(0.0, 0.0);
    GTYPE* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(ret_gpu, &ret, sizeof(GTYPE),
                       gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);
    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(inner_product_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    inner_product_gpu<<<grid, block, 0, *gpu_stream>>>(
        ret_gpu, bra_state_gpu, ket_state_gpu, dim);

    // Check for any errors launching the kernel
    gpuStatus = gpuGetLastError();

    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&ret, ret_gpu, sizeof(GTYPE),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);

    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    bra_state = reinterpret_cast<const void*>(bra_state_gpu);
    ket_state = reinterpret_cast<const void*>(ket_state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret;
}

__host__ CPPCTYPE inner_product_host(const void* bra_state,
    const void* ket_state, ITYPE dim, void* stream,
    unsigned int device_number) {
    if (dim <= INT_MAX) {
        // あとでcublas版を使うように直す
        return inner_product_original_host(
            bra_state, ket_state, dim, stream, device_number);
        // return inner_product_cublas_host(bra_state, ket_state, dim, stream,
        // device_number);
    } else {
        return inner_product_original_host(
            bra_state, ket_state, dim, stream, device_number);
    }
}

__global__ void expectation_value_PauliI_gpu(
    double* ret, GTYPE* state, unsigned int target_qubit_index, ITYPE dim) {
    double sum = 0.0;
    ITYPE loop_dim = dim;
    GTYPE tmp_state;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        tmp_state = state[state_index];
        sum += gpuCreal(gpuCmul(gpuConj(tmp_state), tmp_state));
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

__global__ void expectation_value_PauliX_gpu(
    double* ret, GTYPE* state, unsigned int target_qubit_index, ITYPE dim) {
    double sum = 0.0;
    ITYPE basis0, basis1;
    ITYPE loop_dim = dim >> 1;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        basis0 = (state_index >> target_qubit_index);
        basis0 = basis0 << (target_qubit_index + 1);
        basis0 += state_index & ((1ULL << target_qubit_index) - 1);
        basis1 = basis0 ^ (1ULL << target_qubit_index);

        sum += gpuCreal(gpuCmul(gpuConj(state[basis0]), state[basis1]));
    }
    sum *= 2;
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

__global__ void expectation_value_PauliY_gpu(
    double* ret, GTYPE* state, unsigned int target_qubit_index, ITYPE dim) {
    double sum = 0.0;
    ITYPE basis0, basis1;
    ITYPE loop_dim = dim >> 1;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        basis0 = (state_index >> target_qubit_index);
        basis0 = basis0 << (target_qubit_index + 1);
        basis0 += state_index & ((1ULL << target_qubit_index) - 1);
        basis1 = basis0 ^ (1ULL << target_qubit_index);
        sum += gpuCimag(gpuCmul(gpuConj(state[basis0]), state[basis1]));
    }
    sum *= 2;
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

__global__ void expectation_value_PauliZ_gpu(
    double* ret, GTYPE* state, unsigned int target_qubit_index, ITYPE dim) {
    double sum = 0.0;
    ITYPE basis0, basis1;
    ITYPE loop_dim = dim >> 1;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        basis0 = (state_index >> target_qubit_index);
        basis0 = basis0 << (target_qubit_index + 1);
        basis0 += state_index & ((1ULL << target_qubit_index) - 1);
        basis1 = basis0 ^ (1ULL << target_qubit_index);
        sum += gpuCreal(gpuCmul(gpuConj(state[basis0]), state[basis0])) -
               gpuCreal(gpuCmul(gpuConj(state[basis1]), state[basis1]));
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

__host__ double expectation_value_single_qubit_Pauli_operator_host(
    unsigned int operator_index, unsigned int target_qubit_index, void* state,
    ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    double h_ret = 0.0;
    double* d_ret;

    checkGpuErrors(
        gpuMalloc((void**)&d_ret, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(d_ret, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    if (operator_index == 1) {
        ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);
        unsigned int max_block_size =
            get_block_size_to_maximize_occupancy(expectation_value_PauliX_gpu);
        unsigned int block =
            loop_dim <= max_block_size ? loop_dim : max_block_size;
        unsigned int grid = (loop_dim + block - 1) / block;
        expectation_value_PauliX_gpu<<<grid, block, 0, *gpu_stream>>>(
            d_ret, state_gpu, target_qubit_index, dim);
    } else if (operator_index == 2) {
        ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);
        unsigned int max_block_size =
            get_block_size_to_maximize_occupancy(expectation_value_PauliY_gpu);
        unsigned int block =
            loop_dim <= max_block_size ? loop_dim : max_block_size;
        unsigned int grid = (loop_dim + block - 1) / block;
        expectation_value_PauliY_gpu<<<grid, block, 0, *gpu_stream>>>(
            d_ret, state_gpu, target_qubit_index, dim);
    } else if (operator_index == 3) {
        ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);
        unsigned int max_block_size =
            get_block_size_to_maximize_occupancy(expectation_value_PauliZ_gpu);
        unsigned int block =
            loop_dim <= max_block_size ? loop_dim : max_block_size;
        unsigned int grid = (loop_dim + block - 1) / block;
        expectation_value_PauliZ_gpu<<<grid, block, 0, *gpu_stream>>>(
            d_ret, state_gpu, target_qubit_index, dim);
    } else if (operator_index == 0) {
        ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);
        unsigned int max_block_size =
            get_block_size_to_maximize_occupancy(expectation_value_PauliI_gpu);
        unsigned int block =
            loop_dim <= max_block_size ? loop_dim : max_block_size;
        unsigned int grid = (loop_dim + block - 1) / block;
        expectation_value_PauliI_gpu<<<grid, block, 0, *gpu_stream>>>(
            d_ret, state_gpu, target_qubit_index, dim);
    } else {
        printf("operator_index must be an integer of 0, 1, 2, or 3!!");
    }

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&h_ret, d_ret, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(d_ret), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return h_ret;
}

__device__ void multi_Z_gate_device(ITYPE bit_mask, ITYPE DIM, GTYPE* psi_gpu) {
    ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int minus_cnt = 0;
    if (idx < DIM) {
        minus_cnt = __popcll(idx & bit_mask);
        if (minus_cnt & 1)
            psi_gpu[idx] =
                make_gpuDoubleComplex(-psi_gpu[idx].x, -psi_gpu[idx].y);
    }
}

__global__ void multi_Z_gate_gpu(ITYPE bit_mask, ITYPE DIM, GTYPE* psi_gpu) {
    multi_Z_gate_device(bit_mask, DIM, psi_gpu);
}

__host__ void multi_Z_gate_host(int* gates, void* state, ITYPE dim,
    int n_qubits, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    ITYPE bit_mask = 0;
    for (int i = 0; i < n_qubits; ++i) {
        if (gates[i] == 3) bit_mask ^= (1ULL << i);
    }
    gpuError_t gpuStatus;
    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(multi_Z_gate_gpu);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;
    multi_Z_gate_gpu<<<grid, block, 0, *gpu_stream>>>(bit_mask, dim, state_gpu);
    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    gpuStatus = gpuGetLastError();
    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
}

__device__ GTYPE multi_Z_get_expectation_value_device(
    ITYPE idx, ITYPE bit_mask, ITYPE dim, GTYPE* psi_gpu) {
    GTYPE ret = make_gpuDoubleComplex(0.0, 0.0);
    // ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int minus_cnt = 0;
    if (idx < dim) {
        GTYPE tmp_psi_gpu = psi_gpu[idx];
        minus_cnt = __popcll(idx & bit_mask);
        if (minus_cnt & 1)
            tmp_psi_gpu = make_gpuDoubleComplex(-tmp_psi_gpu.x, -tmp_psi_gpu.y);
        ret = gpuCmul(gpuConj(psi_gpu[idx]), tmp_psi_gpu);
    }
    return ret;
}

__global__ void multi_Z_get_expectation_value_gpu(
    GTYPE* ret, ITYPE bit_mask, ITYPE DIM, GTYPE* psi_gpu) {
    GTYPE sum = make_gpuDoubleComplex(0.0, 0.0);
    GTYPE tmp;
    for (ITYPE state = blockIdx.x * blockDim.x + threadIdx.x; state < DIM;
         state += blockDim.x * gridDim.x) {
        tmp =
            multi_Z_get_expectation_value_device(state, bit_mask, DIM, psi_gpu);
        sum = gpuCadd(sum, tmp);
    }
    sum.x = warpReduceSum_double(sum.x);
    sum.y = warpReduceSum_double(sum.y);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0].x), sum.x);
        atomicAdd_double(&(ret[0].y), sum.y);
    }
}

__device__ GTYPE multipauli_get_expectation_value_device(
    ITYPE idx, ITYPE DIM, GTYPE* psi_gpu, int n_qubits) {
    GTYPE ret;
    GTYPE tmp_psi, tmp_prev_state_psi, tmp_state_psi;
    ITYPE prev_state, state;
    int num_y1 = 0;
    int num_z1 = 0;
    int i_cnt = 0;
    int minus_cnt = 0;
    if (idx < DIM) {
        i_cnt = num_pauli_op_gpu[2];
        num_y1 = __popcll(idx & bit_mask_gpu[2]);
        num_z1 = __popcll(idx & bit_mask_gpu[3]);
        minus_cnt = num_y1 + num_z1;
        prev_state = idx;
        state = idx ^ (bit_mask_gpu[1] + bit_mask_gpu[2]);
        tmp_prev_state_psi = psi_gpu[prev_state];
        tmp_state_psi = psi_gpu[state];
        // swap
        tmp_psi = tmp_state_psi;
        tmp_state_psi = tmp_prev_state_psi;
        tmp_prev_state_psi = tmp_psi;
        if (minus_cnt & 1)
            tmp_state_psi =
                make_gpuDoubleComplex(-tmp_state_psi.x, -tmp_state_psi.y);
        if (i_cnt & 1)
            tmp_state_psi =
                make_gpuDoubleComplex(tmp_state_psi.y, tmp_state_psi.x);
        if ((i_cnt >> 1) & 1)
            tmp_state_psi =
                make_gpuDoubleComplex(-tmp_state_psi.x, -tmp_state_psi.y);
        // tmp_state      -> state      : state*conj(tmp_state)
        // tmp_prev_state -> prev_state : prev_state*conj(tmp_prev_state)
        ret = gpuCmul(tmp_state_psi, gpuConj(psi_gpu[state]));
    }
    return ret;
}

__global__ void multipauli_get_expectation_value_gpu(
    GTYPE* ret, ITYPE DIM, GTYPE* psi_gpu, int n_qubits) {
    GTYPE sum = make_gpuDoubleComplex(0.0, 0.0);
    GTYPE tmp;
    for (ITYPE state = blockIdx.x * blockDim.x + threadIdx.x; state < DIM;
         state += blockDim.x * gridDim.x) {
        tmp = multipauli_get_expectation_value_device(
            state, DIM, psi_gpu, n_qubits);
        sum = gpuCadd(sum, tmp);
    }
    sum.x = warpReduceSum_double(sum.x);
    sum.y = warpReduceSum_double(sum.y);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0].x), sum.x);
        atomicAdd_double(&(ret[0].y), sum.y);
    }
}

__host__ double multipauli_get_expectation_value_host(unsigned int* gates,
    void* state, ITYPE dim, int n_qubits, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    CPPCTYPE ret[1];
    ret[0] = CPPCTYPE(0, 0);
    GTYPE* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);

    unsigned int num_pauli_op[4] = {0, 0, 0, 0};
    for (int i = 0; i < n_qubits; ++i) ++num_pauli_op[gates[i]];
    ITYPE bit_mask[4] = {0, 0, 0, 0};
    for (int i = 0; i < n_qubits; ++i) {
        bit_mask[gates[i]] ^= (1ULL << i);
    }
    if (num_pauli_op[1] == 0 && num_pauli_op[2] == 0) {
        unsigned int max_block_size = get_block_size_to_maximize_occupancy(
            multi_Z_get_expectation_value_gpu);
        unsigned int block =
            loop_dim <= max_block_size ? loop_dim : max_block_size;
        unsigned int grid = (loop_dim + block - 1) / block;
        multi_Z_get_expectation_value_gpu<<<grid, block, 0, *gpu_stream>>>(
            ret_gpu, bit_mask[3], dim, state_gpu);
        checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
        checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
        checkGpuErrors(gpuMemcpyAsync(ret, ret_gpu, sizeof(CPPCTYPE),
                           gpuMemcpyDeviceToHost, *gpu_stream),
            __FILE__, __LINE__);
        checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
        state = reinterpret_cast<void*>(state_gpu);
        return ret[0].real();
    }

    unsigned int max_block_size = get_block_size_to_maximize_occupancy(
        multipauli_get_expectation_value_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;
    checkGpuErrors(
        gpuMemcpyToSymbolAsync(GPU_SYMBOL(num_pauli_op_gpu), num_pauli_op,
            sizeof(unsigned int) * 4, 0, gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(
        gpuMemcpyToSymbolAsync(GPU_SYMBOL(bit_mask_gpu), bit_mask,
            sizeof(ITYPE) * 4, 0, gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);

    multipauli_get_expectation_value_gpu<<<grid, block, 0, *gpu_stream>>>(
        ret_gpu, dim, state_gpu, n_qubits);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(ret, ret_gpu, sizeof(CPPCTYPE),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret[0].real();
}

// calculate probability with which we obtain 0 at target qubit
__global__ void M0_prob_gpu(
    double* ret, UINT target_qubit_index, const GTYPE* state, ITYPE dim) {
    const ITYPE loop_dim = dim >> 1;
    double sum = 0.;
    double tmp;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_0 =
            insert_zero_to_basis_index_device(state_index, target_qubit_index);
        tmp = gpuCabs(state[basis_0]);
        sum += tmp * tmp;
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

// calculate probability with which we obtain 0 at target qubit
__host__ double M0_prob_host(UINT target_qubit_index, void* state, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    double ret[1] = {0.0};
    double* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);

    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(M0_prob_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    M0_prob_gpu<<<grid, block, 0, *gpu_stream>>>(
        ret_gpu, target_qubit_index, state_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(ret, ret_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret[0];
}

// calculate probability with which we obtain 1 at target qubit
__global__ void M1_prob_gpu(
    double* ret, UINT target_qubit_index, const GTYPE* state, ITYPE dim) {
    const ITYPE loop_dim = dim >> 1;
    const ITYPE mask = 1ULL << target_qubit_index;
    double sum = 0.;
    double tmp;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_1 =
            insert_zero_to_basis_index_device(state_index, target_qubit_index) ^
            mask;
        tmp = gpuCabs(state[basis_1]);
        sum += tmp * tmp;
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0]), sum);
    }
}

__host__ double M1_prob_host(UINT target_qubit_index, void* state, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    double ret[1] = {0.0};
    double* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);
    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(M1_prob_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    M1_prob_gpu<<<grid, block, 0, *gpu_stream>>>(
        ret_gpu, target_qubit_index, state_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(ret, ret_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret[0];
}

// calculate merginal probability with which we obtain the set of values
// measured_value_list at sorted_target_qubit_index_list warning:
// sorted_target_qubit_index_list must be sorted.
__global__ void marginal_prob_gpu(double* ret_gpu,
    const UINT* sorted_target_qubit_index_list, const UINT* measured_value_list,
    UINT target_qubit_index_count, const GTYPE* state, ITYPE dim) {
    ITYPE loop_dim = dim >> target_qubit_index_count;
    double sum = 0.;
    double tmp;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis = state_index;
        for (UINT cursor = 0; cursor < target_qubit_index_count; cursor++) {
            UINT insert_index = sorted_target_qubit_index_list[cursor];
            ITYPE mask = 1ULL << insert_index;
            basis = insert_zero_to_basis_index_device(basis, insert_index);
            basis ^= mask * measured_value_list[cursor];
        }
        tmp = gpuCabs(state[basis]);
        sum += tmp * tmp;
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret_gpu[0]), sum);
    }
}

__host__ double marginal_prob_host(UINT* sorted_target_qubit_index_list,
    UINT* measured_value_list, UINT target_qubit_index_count, void* state,
    ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    double ret[1] = {0.0};
    double* ret_gpu;
    UINT* sorted_target_qubit_index_list_gpu;
    UINT* measured_value_list_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(ret_gpu, ret, sizeof(double),
                       gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuMalloc((void**)&sorted_target_qubit_index_list_gpu,
                       sizeof(UINT) * target_qubit_index_count),
        __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(sorted_target_qubit_index_list_gpu,
                       sorted_target_qubit_index_list,
                       sizeof(UINT) * target_qubit_index_count,
                       gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuMalloc((void**)&measured_value_list_gpu,
                       sizeof(UINT) * target_qubit_index_count),
        __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(measured_value_list_gpu, measured_value_list,
                       sizeof(UINT) * target_qubit_index_count,
                       gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim =
        get_loop_dim_of_reduction_function(dim >> target_qubit_index_count);
    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(marginal_prob_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    marginal_prob_gpu<<<grid, block, 0, *gpu_stream>>>(ret_gpu,
        sorted_target_qubit_index_list_gpu, measured_value_list_gpu,
        target_qubit_index_count, state_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(ret, ret_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    checkGpuErrors(
        gpuFree(sorted_target_qubit_index_list_gpu), __FILE__, __LINE__);
    checkGpuErrors(gpuFree(measured_value_list_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret[0];
}

__global__ void expectation_value_multi_qubit_Pauli_operator_XZ_mask_gpu(
    double* ret_gpu, ITYPE bit_flip_mask, ITYPE phase_flip_mask,
    UINT global_phase_90rot_count, UINT pivot_qubit_index, GTYPE* state,
    ITYPE dim) {
    const ITYPE loop_dim = dim >> 1;
    double sum = 0.;
    double tmp;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_0 =
            insert_zero_to_basis_index_device(state_index, pivot_qubit_index);
        ITYPE basis_1 = basis_0 ^ bit_flip_mask;
        UINT sign_0 = __popcll(basis_0 & phase_flip_mask) & 1;

        tmp =
            gpuCreal(gpuCmul(gpuCmul(state[basis_0], gpuConj(state[basis_1])),
                PHASE_90ROT_gpu[(global_phase_90rot_count + sign_0 * 2) & 3])) *
            2.0;
        sum += tmp;
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret_gpu[0]), sum);
    }
}

__host__ double expectation_value_multi_qubit_Pauli_operator_XZ_mask_host(
    ITYPE bit_flip_mask, ITYPE phase_flip_mask, UINT global_phase_90rot_count,
    UINT pivot_qubit_index, void* state, ITYPE dim, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    gpuError_t gpuStatus;
    double ret;
    double* ret_gpu;
    CPPCTYPE PHASE_90ROT[4] = {CPPCTYPE(1.0, 0.0), CPPCTYPE(0.0, 1.0),
        CPPCTYPE(-1.0, 0.0), CPPCTYPE(0.0, -1.0)};

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(
        gpuMemcpyToSymbolAsync(GPU_SYMBOL(PHASE_90ROT_gpu), PHASE_90ROT,
            sizeof(GTYPE) * 4, 0, gpuMemcpyHostToDevice, *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);
    unsigned int max_block_size = get_block_size_to_maximize_occupancy(
        expectation_value_multi_qubit_Pauli_operator_XZ_mask_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    expectation_value_multi_qubit_Pauli_operator_XZ_mask_gpu<<<grid, block, 0,
        *gpu_stream>>>(ret_gpu, bit_flip_mask, phase_flip_mask,
        global_phase_90rot_count, pivot_qubit_index, state_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    gpuStatus = gpuGetLastError();
    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&ret, ret_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);

    return ret;
}

__global__ void expectation_value_multi_qubit_Pauli_operator_Z_mask_gpu(
    double* ret_gpu, ITYPE phase_flip_mask, const GTYPE* state, ITYPE dim) {
    const ITYPE loop_dim = dim;
    double sum = 0.;
    double tmp;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        UINT bit_parity = __popcll(state_index & phase_flip_mask) & 1;
        int sign = 1 - 2 * bit_parity;
        tmp = gpuCabs(state[state_index]);
        sum += tmp * tmp * sign;
    }
    sum = warpReduceSum_double(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret_gpu[0]), sum);
    }
}

__host__ double expectation_value_multi_qubit_Pauli_operator_Z_mask_host(
    ITYPE phase_flip_mask, void* state, ITYPE dim, void* stream,
    unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    gpuError_t gpuStatus;
    double ret;
    double* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(double), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);
    unsigned int max_block_size = get_block_size_to_maximize_occupancy(
        expectation_value_multi_qubit_Pauli_operator_Z_mask_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    expectation_value_multi_qubit_Pauli_operator_Z_mask_gpu<<<grid, block, 0,
        *gpu_stream>>>(ret_gpu, phase_flip_mask, state_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    gpuStatus = gpuGetLastError();
    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&ret, ret_gpu, sizeof(double),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);

    return ret;
}

__host__ double expectation_value_multi_qubit_Pauli_operator_partial_list_host(
    const UINT* target_qubit_index_list, const UINT* Pauli_operator_type_list,
    UINT target_qubit_index_count, void* state, ITYPE dim, void* stream,
    unsigned int device_number) {
    ITYPE bit_flip_mask = 0;
    ITYPE phase_flip_mask = 0;
    UINT global_phase_90rot_count = 0;
    UINT pivot_qubit_index = 0;
    get_Pauli_masks_partial_list_gsim(target_qubit_index_list,
        Pauli_operator_type_list, target_qubit_index_count, &bit_flip_mask,
        &phase_flip_mask, &global_phase_90rot_count, &pivot_qubit_index);
    double result;
    if (bit_flip_mask == 0) {
        result = expectation_value_multi_qubit_Pauli_operator_Z_mask_host(
            phase_flip_mask, state, dim, stream, device_number);
    } else {
        result = expectation_value_multi_qubit_Pauli_operator_XZ_mask_host(
            bit_flip_mask, phase_flip_mask, global_phase_90rot_count,
            pivot_qubit_index, state, dim, stream, device_number);
    }
    return result;
}

__host__ double expectation_value_multi_qubit_Pauli_operator_whole_list_host(
    const UINT* Pauli_operator_type_list, UINT qubit_count, void* state,
    ITYPE dim, void* stream, unsigned int device_number) {
    ITYPE bit_flip_mask = 0;
    ITYPE phase_flip_mask = 0;
    UINT global_phase_90rot_count = 0;
    UINT pivot_qubit_index = 0;
    get_Pauli_masks_whole_list_gsim(Pauli_operator_type_list, qubit_count,
        &bit_flip_mask, &phase_flip_mask, &global_phase_90rot_count,
        &pivot_qubit_index);
    double result;
    if (bit_flip_mask == 0) {
        result = expectation_value_multi_qubit_Pauli_operator_Z_mask_host(
            phase_flip_mask, state, dim, stream, device_number);
    } else {
        result = expectation_value_multi_qubit_Pauli_operator_XZ_mask_host(
            bit_flip_mask, phase_flip_mask, global_phase_90rot_count,
            pivot_qubit_index, state, dim, stream, device_number);
    }
    return result;
}

__global__ void transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_gpu(
    GTYPE* ret_gpu, ITYPE bit_flip_mask, ITYPE phase_flip_mask,
    UINT global_phase_90rot_count, UINT pivot_qubit_index,
    const GTYPE* state_bra, const GTYPE* state_ket, ITYPE dim) {
    const ITYPE loop_dim = dim >> 1;

    GTYPE sum = make_gpuDoubleComplex(0.0, 0.0);
    GTYPE tmp;
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_0 =
            insert_zero_to_basis_index_device(state_index, pivot_qubit_index);
        ITYPE basis_1 = basis_0 ^ bit_flip_mask;

        UINT sign_0 = __popcll(basis_0 & phase_flip_mask) & 1;
        tmp = gpuCmul(gpuCmul(state_ket[basis_0], gpuConj(state_bra[basis_1])),
            PHASE_90ROT_gpu[(global_phase_90rot_count + sign_0 * 2) & 3]);
        sum = gpuCadd(sum, tmp);

        UINT sign_1 = __popcll(basis_1 & phase_flip_mask) & 1;
        tmp = gpuCmul(gpuCmul(state_ket[basis_1], gpuConj(state_bra[basis_0])),
            PHASE_90ROT_gpu[(global_phase_90rot_count + sign_1 * 2) & 3]);
        sum = gpuCadd(sum, tmp);
    }
    sum.x = warpReduceSum_double(sum.x);
    sum.y = warpReduceSum_double(sum.y);
    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret_gpu[0].x), sum.x);
        atomicAdd_double(&(ret_gpu[0].y), sum.y);
    }
}

__host__ CPPCTYPE transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_host(
    ITYPE bit_flip_mask, ITYPE phase_flip_mask, UINT global_phase_90rot_count,
    UINT pivot_qubit_index, void* state_bra, void* state_ket, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_bra_gpu = reinterpret_cast<GTYPE*>(state_bra);
    GTYPE* state_ket_gpu = reinterpret_cast<GTYPE*>(state_ket);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    CPPCTYPE ret;
    GTYPE* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(GTYPE), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim >> 1);
    unsigned int max_block_size = get_block_size_to_maximize_occupancy(
        transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_gpu<<<grid, block,
        0, *gpu_stream>>>(ret_gpu, bit_flip_mask, phase_flip_mask,
        global_phase_90rot_count, pivot_qubit_index, state_bra_gpu,
        state_ket_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&ret, ret_gpu, sizeof(GTYPE),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state_bra = reinterpret_cast<void*>(state_bra_gpu);
    state_ket = reinterpret_cast<void*>(state_ket_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret;
}

__global__ void transition_amplitude_multi_qubit_Pauli_operator_Z_mask_gpu(
    GTYPE* ret, ITYPE phase_flip_mask, GTYPE* state_bra, GTYPE* state_ket,
    ITYPE dim) {
    const ITYPE loop_dim = dim;
    GTYPE sum = make_gpuDoubleComplex(0.0, 0.0);
    for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
         state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        int bit_parity = __popcll(state_index & phase_flip_mask) & 1;
        int sign = 1 - 2 * bit_parity;
        GTYPE tmp =
            gpuCmul(state_ket[state_index], gpuConj(state_bra[state_index]));
        tmp = gpuCmul(make_gpuDoubleComplex((double)sign, 0.0), tmp);
        sum = gpuCadd(sum, tmp);
    }
    sum.x = warpReduceSum_double(sum.x);
    sum.y = warpReduceSum_double(sum.y);

    if ((threadIdx.x & (warpSize - 1)) == 0) {
        atomicAdd_double(&(ret[0].x), sum.x);
        atomicAdd_double(&(ret[0].y), sum.y);
    }
}

__host__ CPPCTYPE transition_amplitude_multi_qubit_Pauli_operator_Z_mask_host(
    ITYPE phase_flip_mask, void* state_bra, void* state_ket, ITYPE dim,
    void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    gpuError_t gpuStatus;
    GTYPE* state_bra_gpu = reinterpret_cast<GTYPE*>(state_bra);
    GTYPE* state_ket_gpu = reinterpret_cast<GTYPE*>(state_ket);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    CPPCTYPE ret;
    GTYPE* ret_gpu;

    checkGpuErrors(
        gpuMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
    checkGpuErrors(gpuMemsetAsync(ret_gpu, 0, sizeof(GTYPE), *gpu_stream),
        __FILE__, __LINE__);

    ITYPE loop_dim = get_loop_dim_of_reduction_function(dim);
    unsigned int max_block_size = get_block_size_to_maximize_occupancy(
        transition_amplitude_multi_qubit_Pauli_operator_Z_mask_gpu);
    unsigned int block = loop_dim <= max_block_size ? loop_dim : max_block_size;
    unsigned int grid = (loop_dim + block - 1) / block;

    transition_amplitude_multi_qubit_Pauli_operator_Z_mask_gpu<<<grid, block, 0,
        *gpu_stream>>>(
        ret_gpu, phase_flip_mask, state_bra_gpu, state_ket_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    gpuStatus = gpuGetLastError();
    checkGpuErrors(gpuStatus, __FILE__, __LINE__);
    checkGpuErrors(gpuMemcpyAsync(&ret, ret_gpu, sizeof(GTYPE),
                       gpuMemcpyDeviceToHost, *gpu_stream),
        __FILE__, __LINE__);
    checkGpuErrors(gpuFree(ret_gpu), __FILE__, __LINE__);
    state_bra = reinterpret_cast<void*>(state_bra_gpu);
    state_ket = reinterpret_cast<void*>(state_ket_gpu);
    // stream = reinterpret_cast<void*>(hip_stream);
    return ret;
}

__host__ CPPCTYPE
transition_amplitude_multi_qubit_Pauli_operator_partial_list_host(
    const UINT* target_qubit_index_list, const UINT* Pauli_operator_type_list,
    UINT target_qubit_index_count, void* state_bra, void* state_ket, ITYPE dim,
    void* stream, unsigned int device_number) {
    ITYPE bit_flip_mask = 0;
    ITYPE phase_flip_mask = 0;
    UINT global_phase_90rot_count = 0;
    UINT pivot_qubit_index = 0;
    get_Pauli_masks_partial_list_gsim(target_qubit_index_list,
        Pauli_operator_type_list, target_qubit_index_count, &bit_flip_mask,
        &phase_flip_mask, &global_phase_90rot_count, &pivot_qubit_index);
    CPPCTYPE result;
    if (bit_flip_mask == 0) {
        result = transition_amplitude_multi_qubit_Pauli_operator_Z_mask_host(
            phase_flip_mask, state_bra, state_ket, dim, stream, device_number);
    } else {
        result = transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_host(
            bit_flip_mask, phase_flip_mask, global_phase_90rot_count,
            pivot_qubit_index, state_bra, state_ket, dim, stream,
            device_number);
    }
    return result;
}

__host__ CPPCTYPE
transition_amplitude_multi_qubit_Pauli_operator_whole_list_host(
    const UINT* Pauli_operator_type_list, UINT qubit_count, void* state_bra,
    void* state_ket, ITYPE dim, void* stream, unsigned int device_number) {
    ITYPE bit_flip_mask = 0;
    ITYPE phase_flip_mask = 0;
    UINT global_phase_90rot_count = 0;
    UINT pivot_qubit_index = 0;
    get_Pauli_masks_whole_list_gsim(Pauli_operator_type_list, qubit_count,
        &bit_flip_mask, &phase_flip_mask, &global_phase_90rot_count,
        &pivot_qubit_index);
    CPPCTYPE result;
    if (bit_flip_mask == 0) {
        result = transition_amplitude_multi_qubit_Pauli_operator_Z_mask_host(
            phase_flip_mask, state_bra, state_ket, dim, stream, device_number);
    } else {
        result = transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_host(
            bit_flip_mask, phase_flip_mask, global_phase_90rot_count,
            pivot_qubit_index, state_bra, state_ket, dim, stream,
            device_number);
    }
    return result;
}
