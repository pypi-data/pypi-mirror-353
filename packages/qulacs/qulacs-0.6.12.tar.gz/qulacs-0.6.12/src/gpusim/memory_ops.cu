#include "hip/hip_runtime.h"
#include <assert.h>

#include <algorithm>
#include <cmath>
#include <complex>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "gpu_wrapping.h"
#include "memory_ops.h"
#include "memory_ops_device_functions.h"
#include "stat_ops.h"
#include "update_ops_cuda.h"
#include "util.cuh"
#include "util_func.h"
#include "util_type.h"
#include "util_type_internal.h"

__host__ void* allocate_cuda_stream_host(
    unsigned int max_cuda_stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);
    gpuStream_t* stream =
        (gpuStream_t*)malloc(max_cuda_stream * sizeof(gpuStream_t));
    for (unsigned int i = 0; i < max_cuda_stream; ++i)
        gpuStreamCreate(&stream[i]);
    void* gpu_stream = reinterpret_cast<void*>(stream);
    return gpu_stream;
}

__host__ void release_cuda_stream_host(void* gpu_stream,
    unsigned int max_cuda_stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);
    gpuStream_t* stream = reinterpret_cast<gpuStream_t*>(gpu_stream);
    for (unsigned int i = 0; i < max_cuda_stream; ++i)
        gpuStreamDestroy(stream[i]);
    free(stream);
}

__global__ void init_qstate(GTYPE* state_gpu, ITYPE dim) {
    ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim) {
        state_gpu[idx] = make_gpuDoubleComplex(0.0, 0.0);
    }
    if (idx == 0) state_gpu[idx] = make_gpuDoubleComplex(1.0, 0.0);
}

__host__ void* allocate_quantum_state_host(
    ITYPE dim, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);
    GTYPE* state_gpu;
    checkGpuErrors(
        gpuMalloc((void**)&state_gpu, dim * sizeof(GTYPE)), __FILE__, __LINE__);
    void* psi_gpu = reinterpret_cast<void*>(state_gpu);
    return psi_gpu;
}

__host__ void initialize_quantum_state_host(
    void* state, ITYPE dim, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);

    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(init_qstate);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;
    init_qstate<<<grid, block, 0, *gpu_stream>>>(state_gpu, dim);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);
}

__host__ void release_quantum_state_host(
    void* state, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);
    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    checkGpuErrors(gpuFree(state_gpu), __FILE__, __LINE__);
}

__global__ void init_rnd(
    gpurandState* const rnd_state, const unsigned int seed, ITYPE dim) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < dim) {
        gpurand_init(seed, tid, 0, &rnd_state[tid]);
    }
}

/*
__global__ void rand_normal_mtgp32(hiprandState* rnd_state, GTYPE* state, ITYPE
dim){ ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x; double2 rnd;
    hiprandStateMtgp32 localState = rnd_state[idx];
        if (idx < dim) {
        rnd = hiprand_normal2_double(&localState);
        state[idx] = make_hipDoubleComplex(rnd.x, rnd.y);
        rnd_state[idx] = localState;
    }
}
*/

__global__ void rand_normal_xorwow(
    gpurandState* rnd_state, GTYPE* state, ITYPE dim) {
    ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    // double2 rnd;
    double tmp1, tmp2;
    double real, imag;
    gpurandState localState = rnd_state[idx];
    if (idx < dim) {
        // rnd = hiprand_normal2_double(&localState);
        tmp1 = gpurand_uniform_double(&localState);
        tmp2 = gpurand_uniform_double(&localState);
        real = sqrt(-1.0 * log(tmp1)) * sinpi(2.0 * tmp2);
        tmp1 = gpurand_uniform_double(&localState);
        tmp2 = gpurand_uniform_double(&localState);
        imag = sqrt(-1.0 * log(tmp1)) * sinpi(2.0 * tmp2);
        state[idx] = make_gpuDoubleComplex(real, imag);
        rnd_state[idx] = localState;
    }
}

__host__ void initialize_Haar_random_state_with_seed_host(void* state,
    ITYPE dim, UINT seed, void* stream, unsigned int device_number) {
    int current_device = get_current_device();
    if (device_number != current_device) gpuSetDevice(device_number);

    GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
    gpuStream_t* gpu_stream = reinterpret_cast<gpuStream_t*>(stream);
    // const ITYPE ignore_first = 40;
    double norm = 0.;

    gpurandState* rnd_state;
    checkGpuErrors(gpuMalloc((void**)&rnd_state, dim * sizeof(gpurandState)),
        __FILE__, __LINE__);

    // HIPRAND_RNG_PSEUDO_XORWOW
    // HIPRAND_RNG_PSEUDO_MT19937 offset cannot be used and need sm_35 or higher.

    unsigned int max_block_size =
        get_block_size_to_maximize_occupancy(init_rnd);
    unsigned int block = dim <= max_block_size ? dim : max_block_size;
    unsigned int grid = (dim + block - 1) / block;

    init_rnd<<<grid, block, 0, *gpu_stream>>>(rnd_state, seed, dim);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);

    rand_normal_xorwow<<<grid, block, 0, *gpu_stream>>>(
        rnd_state, state_gpu, dim);
    checkGpuErrors(gpuGetLastError(), __FILE__, __LINE__);

    checkGpuErrors(gpuStreamSynchronize(*gpu_stream), __FILE__, __LINE__);
    checkGpuErrors(gpuFree(rnd_state), __FILE__, __LINE__);
    state = reinterpret_cast<void*>(state_gpu);

    norm = state_norm_squared_host(state, dim, gpu_stream, device_number);
    normalize_host(norm, state, dim, gpu_stream, device_number);
}

__host__ void initialize_Haar_random_state_host(
    void* state, ITYPE dim, void* stream, unsigned int device_number) {
    initialize_Haar_random_state_with_seed_host(
        state, dim, (unsigned)time(NULL), stream, device_number);
}
